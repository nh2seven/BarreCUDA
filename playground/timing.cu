#include <iostream>
#include <hip/hip_runtime.h>

__global__ void dummy_kernel(float *out)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    out[idx] = idx * 2.0f;
}

int main()
{
    const int N = 1024;
    float *d_out;
    hipMalloc(&d_out, N * sizeof(float));

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start and stop events around the kernel launch
    hipEventRecord(start);
    dummy_kernel<<<4, 256>>>(d_out);
    hipEventRecord(stop);

    // Wait for the kernel to finish
    hipEventSynchronize(stop);

    // Calculate the elapsed time
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    std::cout << "Kernel time: " << ms << " ms" << std::endl;

    // Clean up
    hipFree(d_out);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}